#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include "error_check.hpp"
#include "sampling.h"

/**
 * @todo
 * 1. output不够完整，应该输出9维向量，（位置，方向，RGB），RGB直接在此处输出是为了减少外部变形concatenate的时间
 * 2. 写两个接口，一个是RGB放在最后（6维，最后一个向量前三维度是RGB，后三为0），另一个是RGB在每一个向量中
 * 3. 后一个接口的RGB也需要放在shared memory中
 * 4. [... python是什么意思？] （run_nerf.py 202）
 * 5. 个人感觉，只有方向是需要normalized的，一时因为origin normalize之后就没有意义了，二是官方实现中除了NDC之外貌似也没有对origin进行显式normalize
 * 6. 方向的normalize是l2 norm
 */

/// offset is used in multi-stream concurrency
/// params is (ray_num, 3, 4)
__global__ void getSampledPoints(
    const float *const imgs, const float* const params, float *output, float *lengths,
    hiprandState *r_state, int cam_num, int width, int height, int offset, float near_t, float resolution)
{
    extern __shared__ float transforms[];           /// 9 floats for RK^{-1}, 3 floats for t, 1 int for sampled_id 
    int* sample_id = (int*)(transforms + 12);
    const int ray_id = blockIdx.x + offset, bin_id = threadIdx.x, bin_num = blockDim.x, image_size = width * height;
    const int state_id = ray_id * bin_num + bin_id;
    int cam_id = 0, row_id = 0, col_id = 0, id_in_img = 0;
    /// copy PK^-1 from global memory to shared local memory, enabling faster accessing
    hiprand_init(state_id, 0, 0, &r_state[state_id]);
    if (bin_id == 0) {
        *sample_id = hiprand(&r_state[state_id]) % (cam_num * image_size);
        cam_id = *sample_id / (image_size);
        const float* const ptr = params + 12 * cam_id;
        for (int i = 0; i < 12; i++)
            transforms[i] = ptr[i];
        // printf("%d, %d, [%d, %d]\n", id_in_img / width, id_in_img % width, *sample_id, ray_id);
    }
    __syncthreads();
    id_in_img = (*sample_id % image_size);
    cam_id = *sample_id / (image_size), row_id = id_in_img / width, col_id = id_in_img % width;
    Eigen::Matrix3f T;       // A is equal to PK^-1, these are ex(in)trinsics respectively
    Eigen::Vector3f t;
    T << transforms[0], transforms[1], transforms[2], transforms[4], transforms[5], transforms[6], transforms[8], transforms[9], transforms[10];
    t << transforms[3], transforms[7], transforms[11];
    Eigen::Vector3f raw_dir = T * Eigen::Vector3f(col_id, row_id, 1.0);
    raw_dir = (raw_dir / raw_dir.norm()).eval();            // normalized direction in world frame
    float sample_depth = near_t + resolution * bin_id + hiprand_uniform(&r_state[state_id]) * resolution;
    const int ray_base = ray_id * bin_num, total_base = (ray_base + ray_id + bin_id) * 3;
    lengths[ray_base + bin_id] = sample_depth;
    Eigen::Vector3f p = raw_dir * sample_depth + t;
    output[total_base] = p.x();
    output[total_base + 1] = p.y();
    output[total_base + 2] = p.z();
    if (bin_id == 0) {
        const int image_offset = row_id * width + col_id, batch_base = 3 * image_size * cam_id, rgb_base = (ray_base + ray_id + bin_num) * 3;
        output[rgb_base] = imgs[batch_base + image_offset];
        output[rgb_base + 1] = imgs[batch_base + image_size + image_offset];
        output[rgb_base + 2] = imgs[batch_base + image_size << 1 + image_offset];
    }
    __syncthreads();
}

/// input tensor imgs (N, 3, H, W),
/// camera poses, which should be convert to Eigen, the shape is (batch (number of cams), 3, 4)
/// output: 1. (sample_ray_num, sample_bin_num + 1, 3), points sampled and the gt color 2. length (sample_ray_num, sample_bin_num)
/// both of the output is a tensor
__host__ void cudaSamplerKernel(
    at::Tensor imgs, at::Tensor tfs, at::Tensor output, at::Tensor lengths,
    int sample_ray_num, int sample_bin_num, float near_t, float resolution
) {
    hiprandState *rand_states;
    const int batch_size = imgs.size(0), width = imgs.size(3), height = imgs.size(2);
    CUDA_CHECK_RETURN(hipMalloc((void **)&rand_states, sample_ray_num * sample_bin_num * sizeof(hiprandState)));
    const float* const img_data = imgs.data_ptr<float>();
    const float* const param_data = tfs.data_ptr<float>();
    float* output_data = output.data_ptr<float>();
    float* length_data = lengths.data_ptr<float>();

    /// GPU stream concurrency
    hipStream_t streams[8];
    for (int i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    /// make sure that number of rays to sample is the multiple of 16
    int cascade_num = sample_ray_num >> 4;      // sample_ray_num / 16
    for (int i = 0; i < cascade_num; i++) {
        getSampledPoints <<< 16, sample_bin_num, 13, streams[i % 8]>>> (
            img_data, param_data, output_data, length_data, rand_states, batch_size, width, height, i << 4, near_t, resolution
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipFree(rand_states));
}
