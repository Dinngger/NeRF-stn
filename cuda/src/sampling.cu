#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include "error_check.hpp"
#include "sampling.h"

/// grid中有 sample number 个 block(M)，每个block有 number of samples per ray(N)个线程
/// output已经初始化为(M, N+1, 3), lengths为(M, N)
/// offset is used in multi-stream concurrency
__global__ void getSampledPoints(
    const float *const imgs, const float* const params, float *output, float *lengths,
    hiprandState *r_state, int cam_num, int width, int height, int offset, float near_t, float resolution)
{
    extern __shared__ float transforms[];           /// 9 floats for R
    int* sample_id = (int*)(transforms + 9);
    const int ray_id = blockIdx.x + offset, bin_id = threadIdx.x, bin_num = blockDim.x, image_size = width * height;
    short cam_id = 0, row_id = 0, col_id = 0, id_in_img = 0, state_id = ray_id * bin_num;
    /// copy PK^-1 from global memory to shared local memory, enabling faster accessing
    if (bin_id == 0) {
        hiprand_init(ray_id, 0, 0, &r_state[ray_id]);
        *sample_id = hiprand(&r_state[state_id]) % (cam_num * image_size);
        cam_id = *sample_id / (image_size);
        const float* const ptr = params + 9 * cam_id;
        for (int i = 0; i < 9; i++)
            transforms[i] = ptr[i];
    }
    __syncthreads();
    id_in_img = (*sample_id % image_size);
    cam_id = *sample_id / (image_size), row_id = id_in_img / width, col_id = id_in_img % width;
    
    Eigen::Matrix3f A;       // A is equal to PK^-1, these are ex(in)trinsics respectively
    A << transforms[0], transforms[1], transforms[2], transforms[3], transforms[4], transforms[5], transforms[6], transforms[7], transforms[8];
    Eigen::Vector3f raw_dir = A * Eigen::Vector3f(col_id, row_id, 1.0);
    raw_dir = (raw_dir / raw_dir.norm()).eval();            // normalized direction in world frame
    float sample_depth = near_t + resolution * bin_id + hiprand_uniform(&r_state[state_id + bin_id]) * resolution;
    const int ray_base = ray_id * bin_num, total_base = (ray_base + ray_id + bin_id) * 3;
    lengths[ray_base + bin_id] = sample_depth;
    Eigen::Vector3f p = raw_dir * sample_depth;
    output[total_base] = p.x();
    output[total_base + 1] = p.y();
    output[total_base + 2] = p.z();
    if (bin_id == 0) {
        const int image_offset = row_id * width + col_id, batch_base = 3 * image_size * cam_id, rgb_base = (ray_base + ray_id + bin_num) * 3;
        output[rgb_base] = imgs[batch_base + image_offset];
        output[rgb_base + 1] = imgs[batch_base + image_size + image_offset];
        output[rgb_base + 2] = imgs[batch_base + image_size << 1 + image_offset];
    }
    __syncthreads();
}

/// input tensor imgs (N, 3, H, W),
/// camera poses, which should be convert to Eigen, the shape is (batch (number of cams), 3, 3)
/// output: 1. (sample_ray_num, sample_bin_num + 1, 3), points sampled and the gt color 2. length (sample_ray_num, sample_bin_num)
/// both of the output is a tensor
__host__ void cudaSamplerKernel(
    at::Tensor imgs, at::Tensor tfs, at::Tensor output, at::Tensor lengths,
    int sample_ray_num, int sample_bin_num, float near_t, float resolution
) {
    hiprandState *rand_states;
    const int batch_size = imgs.size(0), width = imgs.size(3), height = imgs.size(2);
    CUDA_CHECK_RETURN(hipMalloc((void **)&rand_states, sample_ray_num * sample_bin_num * sizeof(hiprandState)));

    const float* const img_data = imgs.data_ptr<float>();
    const float* const param_data = tfs.data_ptr<float>();
    float* output_data = output.data_ptr<float>();
    float* length_data = lengths.data_ptr<float>();

    /// GPU stream concurrency
    hipStream_t streams[8];
    for (int i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    /// make sure that number of rays to sample is the multiple of 16
    int cascade_num = sample_ray_num >> 4;      // sample_ray_num / 16
    for (int i = 0; i < cascade_num; i++) {
        getSampledPoints <<< 16, sample_bin_num, 10, streams[i % 8]>>> (
            img_data, param_data, output_data, length_data, rand_states, batch_size, width, height, i << 4, near_t, resolution
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipFree(rand_states));
}
